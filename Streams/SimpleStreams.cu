///////////////////////////////////////////////////////
/////////
/////////  Simple program to explain streams usage in CUDA
/////////  
/////////  
/////////	Author: Francesco Caruso
/////////	francescocaruso979@gmail.com


#include <iostream>

#include <hip/hip_runtime.h>

////////////////////////
/// Error Handling code
////////////////////////

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

//--------------------------------------------------------------------------

int main(int argc, char* argv[])
{
	int whichDevice = 0;
	hipDeviceProp_t prop;
	
	HANDLE_ERROR( hipSetDevice(whichDevice) );
	
	HANDLE_ERROR( hipGetDeviceProperties( &prop, whichDevice );

	return 0;
}