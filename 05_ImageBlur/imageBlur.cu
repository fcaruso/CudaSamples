#include "hip/hip_runtime.h"
#include <iostream>

// CUDA utilities and system includes
#include <hip/hip_runtime.h>

// Helper functions
#include <helper_functions.h>  // CUDA SDK Helper functions
#include <hip/hip_runtime_api.h>       // CUDA device initialization helper functions
#include <helper_image.h>

#include <hip/hip_runtime_api.h>

char *image_filename = "C:/ProgramData/NVIDIA Corporation/CUDA Samples/v6.5/common/data/lena.pgm";
unsigned int width, height;
unsigned char *h_img  = NULL;
unsigned char *d_img  = NULL;

#define BLOCK_WIDTH		32
#define BLOCK_HEIGHT	32

template<unsigned short RADIUS >
__global__ void kRadialBlur( ... )
{

			
}

__global__ void kBlur( ... )
{


}

int main(int argc, char* argv[])
{
	hipProfilerStart();
    // load image (needed so we can get the width and height before we create the window
	sdkLoadPGM(image_filename, (unsigned char **) &h_img, &width, &height);
	printf("width: %d \t height: %d \n", width, height);

	// fill GPU 
	unsigned char* d_img = NULL;
	size_t pitch;
	hipMallocPitch( .... );
	hipMemcpy2D( .... );

	// process image
	dim3 dGrid( ... );
	dim3 dBlock( ... );
	kBlur<<<....>>> ( .... )
	kRadialBlur<4> <<< .... >>> ( .... );
	hipDeviceSynchronize();
	// save image
	hipMemcpy2D( .... );
	sdkSavePGM("blurred.ppm", h_img, width, height );

	// free memory
	hipFree( d_img );
	hipProfilerStop();
	return 0;
}